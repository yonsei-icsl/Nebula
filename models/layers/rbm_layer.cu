#include "hip/hip_runtime.h"
extern "C++" {
#include "rbm_layer.h"
#include "utils.h"
#include "gemm.h"
#include "connected_layer.h"
}
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "activations.cu"

namespace nebula {

__global__ void _calc_hidden_bias_update_(float *m_hidden_bias_update_dev, float *m_hidden_mean_zero_step_dev, float *m_hidden_mean_k_step_dev, unsigned m_batch_size, unsigned m_output_size) {
    size_t j = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if(j >=m_output_size) { return; }
    for(unsigned i = 0; i < m_batch_size; i++)
    {
        m_hidden_bias_update_dev[j] += m_hidden_mean_zero_step_dev[i * m_output_size + j] - m_hidden_mean_k_step_dev[i * m_output_size + j];
    }
}

__global__ void _calc_visible_bias_update_(float *m_visible_bias_update_dev, float *m_visible_units_zero_step_dev, float *m_visible_units_k_step_dev, unsigned m_batch_size, unsigned m_input_size) {
    size_t k = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if(k >=m_input_size) { return; }
    
    for(unsigned i = 0; i < m_batch_size; i++)
    {
        m_visible_bias_update_dev[k] += m_visible_units_zero_step_dev[i * m_input_size + k] - m_visible_units_k_step_dev[i * m_input_size + k];
    }
}

extern "C++" void rbm_layer_t::_sample_hidden_units_(unsigned m_step) {
    hipMemset(hidden_units_dev, 0.0, output_size * network->batch_size * sizeof(float));
    
    float *t_visible_units_dev;
    float *t_hidden_mean_dev;

    if(m_step==0) {
        t_visible_units_dev = visible_units_zero_step_dev;
        t_hidden_mean_dev   = hidden_mean_zero_step_dev;
    }
    else {
        t_visible_units_dev = visible_units_k_step_dev;
        t_hidden_mean_dev   = hidden_mean_k_step_dev; 
    }
    
    const float alpha = 1.0;
    const float beta  = 1.0;
    
#ifdef CUSTOM_BLAS  
    _gemm_(HIPBLAS_OP_T, HIPBLAS_OP_N,
           output_size, network->batch_size, input_size, 
           alpha, 
           weight_dev, input_size, 
           t_visible_units_dev, input_size, 
           beta, 
           t_hidden_mean_dev, output_size);

#else
    hipblasSgemm(network->cublas_handle, HIPBLAS_OP_T, HIPBLAS_OP_N, 
                output_size, network->batch_size, input_size, 
                &alpha, 
                weight_dev, input_size, 
                t_visible_units_dev, input_size, 
                &beta,
                t_hidden_mean_dev, output_size);
    
#endif

    _forward_bias_(t_hidden_mean_dev, hidden_bias_dev, 1, output_size, network->batch_size);
    _logistic_activation_(t_hidden_mean_dev, output_size * network->batch_size); 

    hiprandGenerateUniform(network->generator, hidden_units_dev, output_size * network->batch_size);
    _sampling_(hidden_units_dev, t_hidden_mean_dev, output_size * network->batch_size);
}

extern "C++" void rbm_layer_t::_sample_visible_units_() {
    hipMemset(visible_units_k_step_dev, 0.0, input_size * network->batch_size * sizeof(float));

    const float alpha = 1.0;
    const float beta  = 1.0;

#ifdef CUSTOM_BLAS
  
    _gemm_(HIPBLAS_OP_N, HIPBLAS_OP_N,
           input_size, network->batch_size, output_size, 
           alpha, 
           weight_dev, input_size, 
           hidden_units_dev, output_size, 
           beta, 
           visible_mean_dev, input_size);
   
#else
    hipblasSgemm(network->cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 
                input_size, network->batch_size, output_size, 
                &alpha, 
                weight_dev, input_size, 
                hidden_units_dev, output_size, 
                &beta,
                visible_mean_dev, input_size);    
#endif
    
    _forward_bias_(visible_mean_dev, visible_bias_dev, 1, input_size, network->batch_size);
    _logistic_activation_(visible_mean_dev, input_size * network->batch_size); 
    
    hiprandGenerateUniform(network->generator, visible_units_k_step_dev, input_size * network->batch_size);
    _sampling_(visible_units_k_step_dev, visible_mean_dev, input_size * network->batch_size);
}

extern "C++" void rbm_layer_t::_pretrain_() {
	hipMemset(output_data_dev, 0.0, output_size * network->batch_size * sizeof(float));
    hipMemset(delta_dev, 0.0, output_size * network->batch_size * sizeof(float));

    float *input_data_dev = prev_layer ? prev_layer->output_data_dev : network->input_data_dev;
     
    hipMemcpy(visible_units_zero_step_dev, input_data_dev, input_size * network->batch_size * sizeof(float), hipMemcpyDeviceToDevice);

    // K-step contrastive divergence_gradient approximation for weight update and bias update
    for(unsigned t = 0; t < k_step; t++) {
        _sample_hidden_units_(t);
        _sample_visible_units_();
    }
    _sample_hidden_units_(1);
    
    const float alpha  = 1.0;
    const float alpha2 = -1.0;
    const float beta   = 1.0;
    
#ifdef CUSTOM_BLAS  
    // Matrix multiplication for weight update.
    _gemm_(HIPBLAS_OP_N, HIPBLAS_OP_T, 
           input_size, output_size, network->batch_size, 
           alpha, 
           visible_units_zero_step_dev, input_size, 
           hidden_mean_zero_step_dev, output_size,
           beta,
           weight_update_dev, input_size);

    _gemm_(HIPBLAS_OP_N, HIPBLAS_OP_T,
           input_size, output_size, network->batch_size,
           alpha2,
           visible_units_k_step_dev, input_size,
           hidden_mean_k_step_dev, output_size,
           beta, 
           weight_update_dev, input_size);
#else
    // Matrix multiplication for weight update.
    hipblasSgemm(network->cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_T, 
                input_size, output_size, network->batch_size, 
                &alpha, 
                visible_units_zero_step_dev, input_size, 
                hidden_mean_zero_step_dev, output_size, 
                &beta, 
                weight_update_dev, input_size);
    hipblasSgemm(network->cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_T, 
                input_size, output_size, network->batch_size, 
                &alpha2, 
                visible_units_k_step_dev, input_size,
                hidden_mean_k_step_dev, output_size, 
                &beta, 
                weight_update_dev, input_size);
#endif

    // Update hidden bias
    _update_bias_unit_(hidden_bias_update_dev, hidden_mean_zero_step_dev, hidden_mean_k_step_dev, output_size, network->batch_size);
    // and visible bias
    _update_bias_unit_(visible_bias_update_dev, visible_units_zero_step_dev, visible_units_k_step_dev, input_size, network->batch_size);
    
    float learning_rate = network->learning_rate/network->batch_size;
    float decay = (0.0 - network->decay) * network->batch_size;
    float momentum = network->momentum;

#ifdef CUSTOM_BLAS
    _axpy_(input_size, learning_rate, visible_bias_update_dev, 1, visible_bias_dev, 1);
    _scal_(input_size, momentum, visible_bias_update_dev, 1);

    _axpy_(output_size, learning_rate, hidden_bias_update_dev, 1, hidden_bias_dev, 1);
    _scal_(output_size, momentum, hidden_bias_update_dev, 1);

    _axpy_(weight_size, decay, weight_dev, 1, weight_update_dev, 1);
    _axpy_(weight_size, learning_rate, weight_update_dev, 1, weight_dev, 1);
    _scal_(weight_size, momentum, weight_update_dev, 1);
#else
    // Update bias of visible units.
    hipblasSaxpy(network->cublas_handle, input_size, &learning_rate, 
                visible_bias_update_dev, 1, visible_bias_dev, 1);
    hipblasSscal(network->cublas_handle, input_size, &momentum, visible_bias_update_dev, 1);

    // Update bias of hidden units.
    hipblasSaxpy(network->cublas_handle, output_size, &learning_rate, 
                hidden_bias_update_dev, 1, hidden_bias_dev, 1);
    hipblasSscal(network->cublas_handle, output_size, &momentum, hidden_bias_update_dev, 1);

    // Update weight.
    hipblasSaxpy(network->cublas_handle, weight_size, &decay,
                weight_dev, 1, weight_update_dev, 1);
    hipblasSaxpy(network->cublas_handle, weight_size, &learning_rate, 
                weight_update_dev, 1, weight_dev, 1);
    hipblasSscal(network->cublas_handle, weight_size, &momentum, weight_update_dev, 1);
#endif

    hipMemcpy(output_data_dev, hidden_units_dev, output_size * network->batch_size * sizeof(float), hipMemcpyDeviceToDevice);
}

// Forward propagation
extern "C++" void rbm_layer_t::_forward_() {
    hipMemset(output_data_dev, 0, output_size*network->batch_size*sizeof(float));
    hipMemset(delta_dev, 0, output_size*network->batch_size*sizeof(float));

    const float alpha = 1.0;
    const float beta  = 1.0;
    float *input_data_dev = prev_layer ? prev_layer->output_data_dev : network->input_data_dev;

#ifdef QUANTIZATION
    hipMemcpy(weight, weight_dev, 
               weight_size*sizeof(float), hipMemcpyDeviceToHost);
    quantization(weight, DATA_BIT, weight_size, step_size, false);
    hipMemcpy(weight_dev, weight, 
               weight_size*sizeof(float), hipMemcpyHostToDevice);
#endif

#ifdef CUSTOM_BLAS
  
    _gemm_(HIPBLAS_OP_T, HIPBLAS_OP_N,
           output_size, network->batch_size, input_size, 
           alpha, 
           weight_dev, input_size, 
           input_data_dev, input_size, 
           beta, 
           output_data_dev, output_size);
   
#else
    hipblasSgemm(network->cublas_handle, HIPBLAS_OP_T, HIPBLAS_OP_N, 
                output_size, network->batch_size, input_size, 
                &alpha, 
                weight_dev, input_size, 
                input_data_dev, input_size, 
                &beta,
                output_data_dev, output_size);
#endif
    // Forward bias
    _forward_bias_(output_data_dev, hidden_bias_dev, 1, output_size, network->batch_size);
    
    // Activate function
    _activate_();
}

// Backward propagation
extern "C++" void rbm_layer_t::_backward_() {
    // Gradient function 
    _gradient_();
     
    // backward bias.
    _backward_bias_(hidden_bias_update_dev, delta_dev, 1, output_size, network->batch_size);

    const float alpha = 1.0;
    const float beta  = 1.0;
    float *input_data_dev = prev_layer ? prev_layer->output_data_dev : network->input_data_dev;
    float *prev_delta_dev = prev_layer ? prev_layer->delta_dev : NULL;
    
    // Weight update
#ifdef CUSTOM_BLAS
    _gemm_(HIPBLAS_OP_N, HIPBLAS_OP_T, 
           input_size, output_size, network->batch_size, 
           alpha, 
           input_data_dev, input_size, 
           delta_dev, output_size, 
           beta, 
           weight_update_dev, input_size);
#else
    hipblasSgemm(network->cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_T, 
                input_size, output_size, network->batch_size, 
                &alpha, 
                input_data_dev, input_size, 
                delta_dev, output_size, 
                &beta, 
                weight_update_dev, input_size);
#endif
    // Delta update
    if(prev_delta_dev) {
#ifdef CUSTOM_BLAS
        _gemm_(HIPBLAS_OP_N, HIPBLAS_OP_N, 
               input_size, network->batch_size, output_size, 
               alpha, 
               weight_dev, input_size,
               delta_dev, output_size, 
               beta, 
               prev_delta_dev, input_size);
#else
        hipblasSgemm(network->cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 
                    input_size, network->batch_size, output_size, 
                    &alpha, 
                    weight_dev, input_size,
                    delta_dev, output_size, 
                    &beta, 
                    prev_delta_dev, input_size);
#endif
    }
}

extern "C++" void rbm_layer_t::_update_() {
    float learning_rate = network->learning_rate/network->batch_size;
    float decay = -network->decay*network->batch_size;
    float momentum = network->momentum;

#ifdef CUSTOM_BLAS
    _axpy_(weight_size, decay, weight_dev, 1, weight_update_dev, 1);
    _axpy_(weight_size, learning_rate, weight_update_dev, 1, weight_dev, 1);
    _scal_(weight_size, momentum, weight_update_dev, 1);

    _axpy_(output_size, learning_rate, hidden_bias_update_dev, 1, hidden_bias_dev, 1);
    _scal_(output_size, momentum, hidden_bias_update_dev, 1);
#else
    // Weight update
    hipblasSaxpy(network->cublas_handle, weight_size, &decay,
                weight_dev, 1, weight_update_dev, 1);
    hipblasSaxpy(network->cublas_handle, weight_size, &learning_rate, 
                weight_update_dev, 1, weight_dev, 1);
    hipblasSscal(network->cublas_handle, weight_size, &momentum, weight_update_dev, 1);

    // Bias update
    hipblasSaxpy(network->cublas_handle, output_size, &learning_rate, 
                hidden_bias_update_dev, 1, hidden_bias_dev, 1);
    hipblasSscal(network->cublas_handle, output_size, &momentum, hidden_bias_update_dev, 1);
#endif

}

}
// End of namespace nebula.
